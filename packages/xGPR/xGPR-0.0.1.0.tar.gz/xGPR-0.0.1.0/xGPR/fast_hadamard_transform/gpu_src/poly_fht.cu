#include "hip/hip_runtime.h"
/*
* Contains functions needed to run FHT for the polynomial kernel on GPU.
* The input array should already live on GPU.
* The Hadamard transforms are performed using functions from float or double
* array_operations.cu, the diagonal matrix multiplication is slightly different
* and so is implemented here.
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "double_array_operations.h"
#include "float_array_operations.h"
#include "poly_fht.h"

#define DEFAULT_THREADS_PER_BLOCK 256



//Performs an elementwise multiplication of a row of a [1,1,P x S] array against the
//[N,M,S] input array. Note that the dimensions must be checked before calling
//-- done by the wrapper -- and that only S elements of the [1, 1, P x S] array are used.
__global__ void floatPolyFHTMultiplyByDiagonalMat(float *cArray, int8_t *rademArray,
			int dim2, int columnStartPosition, int numElements, float normConstant)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int8_t *rVal = rademArray + columnStartPosition + (j & (dim2 - 1));
    
    if (j < numElements)
        cArray[j] = cArray[j] * *rVal * normConstant;
}



//Performs an elementwise multiplication of a row of a [1,1,P x S] array against the
//[N,M,S] input array. Note that the dimensions must be checked before calling
//-- done by the wrapper -- and that only S elements of the [1, 1, P x S] array are used.
__global__ void doublePolyFHTMultiplyByDiagonalMat(double *cArray, int8_t *rademArray,
			int dim2, int columnStartPosition, int numElements, double normConstant)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int8_t *rVal = rademArray + columnStartPosition + (j & (dim2 - 1));
    
    if (j < numElements)
        cArray[j] = cArray[j] * *rVal * normConstant;
}



//This function performs the FHT operation for the polynomial kernel
//when the input is an arrray of floats.
//Note that reshapedX must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria.
//
//All of these arrays are already expected to "live" on GPU.
const char *floatPolyFHTPrep_(int8_t *radem, float *reshapedX, int reshapedDim0, 
                int reshapedDim1, int reshapedDim2, int numFreqs,
                int columnStartPosition, int rowStartPosition){

    int numElements = reshapedDim0 * reshapedDim1 * reshapedDim2;
    int rowOffset;
    //This is the Hadamard normalization constant.
    float normConstant = log2(reshapedDim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / 
                DEFAULT_THREADS_PER_BLOCK;
    
    //Multiply by radem row 1.
    rowOffset = rowStartPosition * 3 * numFreqs;
    floatPolyFHTMultiplyByDiagonalMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        radem + rowOffset, reshapedDim2, columnStartPosition, numElements,
                        normConstant);
    //First H-transform.
    floatCudaHTransform3d_(reshapedX, reshapedDim0, reshapedDim1, reshapedDim2);

    //Multiply by second row of radem.
    rowOffset += numFreqs;
    floatPolyFHTMultiplyByDiagonalMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        radem + rowOffset, reshapedDim2, columnStartPosition,
                        numElements, normConstant);
    //Second H-transform.
    floatCudaHTransform3d_(reshapedX, reshapedDim0, reshapedDim1, reshapedDim2);
        
    //Multiply by third row of radem.
    rowOffset += numFreqs;
    floatPolyFHTMultiplyByDiagonalMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        radem + rowOffset, reshapedDim2, columnStartPosition,
                        numElements, normConstant);
    //Third H-transform.
    floatCudaHTransform3d_(reshapedX, reshapedDim0, reshapedDim1, reshapedDim2);
    //All operations are in place, no need to return anything except a 
    //no error message. TODO: check the cuda kernels for errors and add error
    //handling.
    return "no_error";
}



//This function performs the FHT operation for the polynomial kernel
//when the input is an arrray of doubles.
//Note that reshapedX must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria.
//
//All of these arrays are already expected to "live" on GPU.
const char *doublePolyFHTPrep_(int8_t *radem, double *reshapedX, int reshapedDim0, 
                int reshapedDim1, int reshapedDim2, int numFreqs,
                int columnStartPosition, int rowStartPosition){

    int numElements = reshapedDim0 * reshapedDim1 * reshapedDim2;
    int rowOffset;
    //This is the Hadamard normalization constant.
    double normConstant = log2(reshapedDim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / 
                DEFAULT_THREADS_PER_BLOCK;
    
    //Multiply by radem row 1.
    rowOffset = rowStartPosition * 3 * numFreqs;
    doublePolyFHTMultiplyByDiagonalMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        radem + rowOffset, reshapedDim2, columnStartPosition, numElements,
                        normConstant);
    //First H-transform.
    doubleCudaHTransform3d_(reshapedX, reshapedDim0, reshapedDim1, reshapedDim2);

    //Multiply by second row of radem.
    rowOffset += numFreqs;
    doublePolyFHTMultiplyByDiagonalMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        radem + rowOffset, reshapedDim2, columnStartPosition,
                        numElements, normConstant);
    //Second H-transform.
    doubleCudaHTransform3d_(reshapedX, reshapedDim0, reshapedDim1, reshapedDim2);
        
    //Multiply by third row of radem.
    rowOffset += numFreqs;
    doublePolyFHTMultiplyByDiagonalMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        radem + rowOffset, reshapedDim2, columnStartPosition,
                        numElements, normConstant);
    //Third H-transform.
    doubleCudaHTransform3d_(reshapedX, reshapedDim0, reshapedDim1, reshapedDim2);
    //All operations are in place, no need to return anything except a 
    //no error message. TODO: check the cuda kernels for errors and add error
    //handling.
    return "no_error";
}
